#include "hip/hip_runtime.h"
#include "cudamat_kernels.cuh"
#include "float.h"
const int NUM_THREADS = 32;

__device__ void reduceToMax(float* sdata, unsigned int tid){

  //Synchronize threads to share shared memory data
  __syncthreads();

  float mySum = sdata[tid];

  // do reduction in shared mem
  if (NUM_THREADS >= 512) { if (tid < 256) { sdata[tid] = mySum = fmaxf(mySum, sdata[tid + 256]); } __syncthreads(); }
  if (NUM_THREADS >= 256) { if (tid < 128) { sdata[tid] = mySum = fmaxf(mySum, sdata[tid + 128]); } __syncthreads(); }
  if (NUM_THREADS >= 128) { if (tid <  64) { sdata[tid] = mySum = fmaxf(mySum, sdata[tid +  64]); } __syncthreads(); }

  if (NUM_THREADS == 32){
    if (tid < 16)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (NUM_THREADS >=  32) { smem[tid] = mySum = fmaxf(mySum, smem[tid + 16]); }
      if (NUM_THREADS >=  16) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  8]); }
      if (NUM_THREADS >=   8) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  4]); }
      if (NUM_THREADS >=   4) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  2]); }
      if (NUM_THREADS >=   2) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  1]); }
    }
  }
  else
  {
    if (tid < 32)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (NUM_THREADS >=  64) { smem[tid] = mySum = fmaxf(mySum, smem[tid + 32]); }
      if (NUM_THREADS >=  32) { smem[tid] = mySum = fmaxf(mySum, smem[tid + 16]); }
      if (NUM_THREADS >=  16) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  8]); }
      if (NUM_THREADS >=   8) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  4]); }
      if (NUM_THREADS >=   4) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  2]); }
      if (NUM_THREADS >=   2) { smem[tid] = mySum = fmaxf(mySum, smem[tid +  1]); }
    }
  }
}

__device__ void reduceToSumLocal(float* sdata, unsigned int tid)
{

  //Synchronize threads to share shared memory data
  __syncthreads();

  float mySum = sdata[tid];

  // do reduction in shared mem
  if (NUM_THREADS >= 512) { if (tid < 256) { sdata[tid] = mySum = mySum + sdata[tid + 256]; } __syncthreads(); }
  if (NUM_THREADS >= 256) { if (tid < 128) { sdata[tid] = mySum = mySum + sdata[tid + 128]; } __syncthreads(); }
  if (NUM_THREADS >= 128) { if (tid <  64) { sdata[tid] = mySum = mySum + sdata[tid +  64]; } __syncthreads(); }

  if (NUM_THREADS == 32){
    if (tid < 16)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (NUM_THREADS >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
      if (NUM_THREADS >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
      if (NUM_THREADS >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
      if (NUM_THREADS >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
      if (NUM_THREADS >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }
  }
  else
  {
    if (tid < 32)
    {
      // now that we are using warp-synchronous programming (below)
      // we need to declare our shared memory volatile so that the compiler
      // doesn't reorder stores to it and induce incorrect behavior.
      volatile float* smem = sdata;
      if (NUM_THREADS >=  64) { smem[tid] = mySum = mySum + smem[tid + 32]; }
      if (NUM_THREADS >=  32) { smem[tid] = mySum = mySum + smem[tid + 16]; }
      if (NUM_THREADS >=  16) { smem[tid] = mySum = mySum + smem[tid +  8]; }
      if (NUM_THREADS >=   8) { smem[tid] = mySum = mySum + smem[tid +  4]; }
      if (NUM_THREADS >=   4) { smem[tid] = mySum = mySum + smem[tid +  2]; }
      if (NUM_THREADS >=   2) { smem[tid] = mySum = mySum + smem[tid +  1]; }
    }
  }
}

/* ------------------------- Random number generation ------------------------- */

__global__ void kSeedRandom(unsigned int* rndMults, unsigned long long* rndWords, unsigned int seed) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;

  // The initial x is the seed and the initial carry is 1
  unsigned long long rndWord = ((unsigned long long)seed << 32) + 1;
  const unsigned int rndMult = rndMults[idx];
  /*
   * Run the chain for a few steps so that all the streams have a chance
   * to differentiate. They start out generating similar random numbers
   * because all the multipliers are similar.
   */
  for(unsigned int i = 0; i < NUM_RND_BURNIN; i++) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
  }
  rndWords[idx] = rndWord;
}

__global__ void kRandomUniform(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    gData[i] = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
  }
  rndWords[idx] = rndWord;
}

__global__ void kRandomGaussian(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  float rnd1, rnd2, R, T;
  for(unsigned int i = idx; i < numElements; i += 2*NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd1 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd2 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    T = 2 * PI * rnd2;
    R = sqrtf(-2 * __logf(rnd1));
    gData[i] = R * __cosf(T);
    if (i + NUM_RND_STREAMS < numElements)
      gData[i + NUM_RND_STREAMS] = R * __sinf(T);
  }
  rndWords[idx] = rndWord;
}

__global__ void kRandomGaussianDropout(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements, float scale) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  float rnd1, rnd2, R, T;
  for(unsigned int i = idx; i < numElements; i += 2*NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd1 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd2 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    T = 2 * PI * rnd2;
    R = sqrtf(-2 * __logf(rnd1));
    gData[i] *= 1 + scale * R * __cosf(T);
    if (i + NUM_RND_STREAMS < numElements)
      gData[i + NUM_RND_STREAMS] *= 1 + scale * R * __sinf(T);
  }
  rndWords[idx] = rndWord;
}


__global__ void kRandomDropout(unsigned int* rndMults, unsigned long long* rndWords, float* gData, unsigned int numElements, float dropprob, float val, float scale) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    gData[i] = (((__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f) > dropprob) ? (scale * gData[i]) : val;
  }
  rndWords[idx] = rndWord;
}

__global__ void kSampleBernoulli(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    target[i] = ((__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f) < gData[i] ? 1:0;
  }
  rndWords[idx] = rndWord;
}
__global__ void kSampleBernoulliTanh(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    target[i] = ((__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f) < (1.0 + gData[i]) / 2.0 ? 1:0;
  }
  rndWords[idx] = rndWord;
}

__global__ void kSamplePoisson(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    target[i] = gData[i];
  }
  rndWords[idx] = rndWord;
}

__global__ void kSampleGaussian(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements, float mult) {

  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];

  float rnd1, rnd2, R, T;
  for(unsigned int i = idx; i < numElements; i += 2*NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd1 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd2 = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    T = 2 * PI * rnd2;
    R = sqrtf(-2 * __logf(rnd1));
    target[i] = gData[i] + mult * R * __cosf(T);
    if (i + NUM_RND_STREAMS < numElements)
      target[i + NUM_RND_STREAMS] = gData[i + NUM_RND_STREAMS] + mult * R * __sinf(T);
  }
  rndWords[idx] = rndWord;
}

__global__ void kPerturbEnergy(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];
  float rnd;

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    target[i] = gData[i] - __logf( - __logf(rnd));
  }
  rndWords[idx] = rndWord;
}

__global__ void kPerturbProb(unsigned int* rndMults, unsigned long long* rndWords, float* gData, float* target, unsigned int numElements) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned long long rndWord = rndWords[idx];
  const unsigned int rndMult = rndMults[idx];
  float rnd;

  for(unsigned int i = idx; i < numElements; i += NUM_RND_STREAMS) {
    rndWord = rndMult * LOW_BITS(rndWord) + HIGH_BITS(rndWord);
    rnd = (__uint2float_rn(LOW_BITS(rndWord)) + 1.0f) / 4294967296.0f;
    target[i] = - gData[i] /  __logf(rnd);
  }
  rndWords[idx] = rndWord;
}


/* ------------------------- Data copying ------------------------- */

/*
   Copy row slice from source to target. There is a block for every 32x32 chunk being copied.
 */
__global__ void kGetRowSlice(float* source, float* target, int start, int end, int width, int height) {
  const int row = start + blockIdx.x * 32 + threadIdx.x;
  const int start_col = blockIdx.y * 32;
  const int end_col = (start_col + 32 < width) ? start_col + 32: width;
  const int target_height = end - start;
  if (row < end) {
    for (int cur_col = start_col; cur_col < end_col; cur_col++)
      target[cur_col * target_height + row - start] = source[cur_col * height + row];
  }
}

__global__ void kSetRowSlice(float* source, float* target, int start, int end, int width, int height) {
  const int row = start + blockIdx.x * 32 + threadIdx.x;
  const int start_col = blockIdx.y * 32;
  const int end_col = (start_col + 32 < width) ? start_col + 32: width;
  const int source_height = end - start;
  if (row < end) {
    for (int cur_col = start_col; cur_col < end_col; cur_col++)
      target[cur_col * height + row] = source[cur_col * source_height + row - start];
    //source[cur_col * height + row - start] = target[cur_col * target_height + row];
  }
}

__global__ void kTranspose(float *odata, float *idata, int width, int height) {
  __shared__ float block[COPY_BLOCK_SIZE][COPY_BLOCK_SIZE+1];

  // read the matrix tile into shared memory
  unsigned int xIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.x;
  unsigned int yIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.y;

  if((xIndex < width) && (yIndex < height)) {
    unsigned int index_in = yIndex * width + xIndex;

    block[threadIdx.y][threadIdx.x] = idata[index_in];
  }

  __syncthreads();

  // write the transposed matrix tile to global memory
  xIndex = blockIdx.y * COPY_BLOCK_SIZE + threadIdx.x;
  yIndex = blockIdx.x * COPY_BLOCK_SIZE + threadIdx.y;

  if((xIndex < height) && (yIndex < width)) {
    unsigned int index_out = yIndex * height + xIndex;

    odata[index_out] = block[threadIdx.x][threadIdx.y];
  }
}
__global__ void kTransposeBig(float *odata, float *idata, int height, int width) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  int r, c;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    r = i % width;
    c = i / width;
    odata[i] = idata[height * r + c];
  }
}


/* ------------------------- Mathematical operations ------------------------- */

__global__ void kLessThan(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] < mat2[i];
}

__global__ void kLessThanEq(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] <= mat2[i];
}

__global__ void kLessThanScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] < val;
}

__global__ void kLessThanEqScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] <= val;
}

__global__ void kGreaterThan(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] > mat2[i];
}

__global__ void kGreaterThanEq(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] >= mat2[i];
}

__global__ void kGreaterThanScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] > val;
}

__global__ void kGreaterThanEqScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] >= val;
}

__global__ void kUpperBound(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] > mat2[i] ? mat2[i] : mat1[i];
}

__global__ void kLowerBound(float* mat1, float* mat2, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat1[i] < mat2[i] ? mat2[i] : mat1[i];
}

__global__ void kUpperBoundScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] > val ? val:mat[i];
}

__global__ void kLowerBoundScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] < val ? val:mat[i];
}

__global__ void kUpperBoundModScalar(float* mat, float val, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] > val ? val : (mat[i] < -val ? -val : mat[i]);
}

__global__ void kSparseDot(int m, int n, int k, float *data, int* indptr, int* indices, float *dense_data, float* target, float beta, float alpha) {
  const unsigned int row = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int col = blockIdx.y * blockDim.y + threadIdx.y;

  if (row < m && col < n) {
    const int start = indptr[row];
    const int end = indptr[row + 1];
    float sum = 0.f;
    for (int i = start; i < end; i++) {
      sum += data[i]  * dense_data[col * k + indices[i]];
    }
    const int pos = col * m + row;
    target[pos] = alpha * sum + ((beta == 0) ? 0 : beta * target[pos]);
  }
}

__global__ void kSign(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] ? copysignf(1., mat[i]) : 0;
}

__global__ void kApplySin(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = __sinf(mat[i]);
}

__global__ void kApplyCos(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = __cosf(mat[i]);
}

__global__ void kApplySigmoid(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = 1 / (1 + __expf(-mat[i]));
}

__global__ void kApplyTanh(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  float mat_i, exp2x;
  for (unsigned int i = idx; i < len; i += numThreads) {
    mat_i = mat[i];
    exp2x = __expf(2 * mat_i);
    target[i] = 1 - 2 / (exp2x + 1);
  }
}

__global__ void kApplyAbs(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = mat[i] * ((mat[i] > 0) - (mat[i] < 0));
}

__global__ void kApplyLog1PlusExp(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  float mat_i;
  for (unsigned int i = idx; i < len; i += numThreads) {
    mat_i = mat[i];
    if (mat_i > 0)
      target[i] = (__logf(1 + __expf(-mat_i)) + mat_i);
    else
      target[i] = __logf(1 + __expf(mat_i));
  }
}

__global__ void kLog(float* mat, float* target, unsigned int len, float tiny) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = __logf(mat[i] + tiny);
}

__global__ void kSquashRelu(float* mat, float* target, unsigned int len, float lambda) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = 2 / (1 + __expf(-lambda * mat[i])) - 1;
}


__global__ void kExp(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = __expf(mat[i]);
}

__global__ void kCeil(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = ceil(mat[i]);
}

__global__ void kFloor(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = floor(mat[i]);
}

__global__ void kSqrt(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = sqrt(mat[i]);
}

__global__ void kPow(float* mat, float pow, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = powf(mat[i], pow);
}

__global__ void kPowMatrix(float* mat, float* pow, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = powf(mat[i], pow[i]);
}

__global__ void kCrossEntropy(float* mat, float* p, float* target, unsigned int len, float tiny) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = -mat[i] * __logf(p[i] + tiny);
}

__global__ void kCrossEntropyBernoulli(float* mat, float* p, float* target, unsigned int len, float tiny) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads)
    target[i] = -mat[i] * __logf(p[i] + tiny) - (1 - mat[i]) * __logf(1 - p[i] + tiny);
}

__global__ void kCorrectPreds(float* mat, float* p, float* target, unsigned int len, float cutoff) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads)
    target[i] = mat[i] * (p[i] >= cutoff) + (1 - mat[i]) * (p[i] < cutoff);
}

__global__ void kReciprocal(float* mat, float* target, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) target[i] = 1. / mat[i];
}

__global__ void kAddColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] + vec[i % height];
  }
}
__global__ void kAddDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width; i += numThreads) {
    tgtMat[width*i + i] = mat[width*i + i] + val;
  }
}

__global__ void kAddDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width; i += numThreads) {
    tgtMat[width*i + i] = mat[width*i + i] + vec[i];
  }
}

__global__ void kMultDiagonalScalar(float* mat, float val, float* tgtMat, unsigned int width) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width; i += numThreads) {
    tgtMat[width*i + i] = mat[width*i + i] * val;
  }
}

__global__ void kMultDiagonal(float* mat, float* vec, float* tgtMat, unsigned int width) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width; i += numThreads) {
    tgtMat[width*i + i] = mat[width*i + i] * vec[i];
  }
}
__global__ void kAddRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] + vec[i / height];
  }
}

__global__ void kAddColMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] + mult * vec[i % height];
  }
}

__global__ void kAddToEachPixel(float* mat1, float* mat2, float* tgtMat, float mult, unsigned int width, unsigned int height, unsigned int num_pix) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat1[i] + mult * mat2[i % height + height * (i / (height * num_pix))];
  }
}
__global__ void kAddRowMult(float* mat, float* vec, float* tgtMat, float mult, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] + mult * vec[i / height];
  }
}
__global__ void kMultByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] * vec[i % height];
  }
}

__global__ void kDivByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] / vec[i / height];
  }
}

__global__ void kDivByColVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] / vec[i % height];
  }
}

__global__ void kMultByRowVector(float* mat, float* vec, float* tgtMat, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    tgtMat[i] = mat[i] * vec[i / height];
  }
}
__global__ void kAddMultSign(float* a, float* b, unsigned int numEls, float mult) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    a[i] = a[i] + ((b[i] > 0) ? mult : ((b[i] < 0) ? -mult : 0));
  }
}
__global__ void kAdd(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] + b[i];
  }
}

__global__ void kSubtract(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] - b[i];
  }
}

__global__ void kDivide(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] / b[i];
  }
}

__global__ void kMult(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * b[i];
  }
}

__global__ void kCosDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = -a[i] * __sinf(b[i]);
  }
}

__global__ void kSinDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * __cosf(b[i]);
  }
}

__global__ void kLogisticDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * b[i] * (1.0 - b[i]);
  }
}

// target[i] < 0 means don't care.
__global__ void kLogisticGrad(float* mat, float* targets, float* out_grad, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    out_grad[i] = (targets[i] < 0) ? 0 : (mat[i] - targets[i]);
  }
}

__global__ void kLogisticCorrectNormalized(float* mat, float* targets, float* out, unsigned int height, unsigned int width) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < height) {
    float correct = 0;
    float total = 0;
    float p, t;
    for (int i = idx; i < width * height; i += height) {
      p = mat[i];
      t = targets[i];
      correct += (t < 0) ? 0 : (((t >= 0.5 && p >= 0.5) || (t < 0.5 && p < 0.5)) ? 1: 0);
      total += (t < 0) ? 0 : 1;
      __syncthreads();
    }
    out[idx] = (total > 0) ? (correct / total) : 0;
  }
}

__global__ void kTanhDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * (1.0 + b[i]) * (1.0 - b[i]) * 0.5;
  }
}

__global__ void kRectifiedLinearDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * (b[i] > 0 ? 1 : 0);
  }
}

__global__ void kRectifiedLinearSmoothDeriv(float* a, float* b, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] * (1 - __expf(-b[i]));
  }
}

__global__ void kMultScalar(float* mat, float alpha, float* dest, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) {
    dest[i] = alpha * mat[i];
  }
}

__global__ void kAssignScalar(float* dest, float alpha, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) {
    dest[i] = alpha;
  }
}

__global__ void kDivideScalar(float* mat, float alpha, float* dest, unsigned int len) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) {
    dest[i] = mat[i] / alpha;
  }
}

__global__ void kAddScalar(float* a, float alpha, float* dest, unsigned int numEls) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < numEls; i += numThreads) {
    dest[i] = a[i] + alpha;
  }
}


__global__ void kSelectRows(float* source, float* target, float* indices, int nRowIs, int nCols, int nSourceRows){
  __shared__ int sourceRowIndices[32];
  const int startTargetRowI = blockIdx.x * 32;
  const int tid = threadIdx.x;
  const int localNRowIs = min(32, nRowIs-startTargetRowI);

  // cooperatively load 32 row indices
  if (tid < localNRowIs){
    sourceRowIndices[tid] = int(indices[startTargetRowI + tid]);
    if (sourceRowIndices[tid]<0)
      sourceRowIndices[tid] += nSourceRows;
    if (sourceRowIndices[tid]<0 || sourceRowIndices[tid]>=nSourceRows)
      sourceRowIndices[tid] = -1;
  }
  __syncthreads();

  // copy 32 rows
  for (int i=0; i<localNRowIs; i++){
    const int targetRowI = startTargetRowI + i, sourceRowI = sourceRowIndices[i];
    for (int colI=tid; colI<nCols; colI+=32)
      target[targetRowI * nCols + colI] = sourceRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
  }
}

__global__ void kSwapColumns(float* source, float* target, float* indices1, float* indices2, int cols, int width, int height){
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  float temp;
  unsigned int column, row, source_pos, target_pos;
  for (unsigned int i = idx; i < height * cols; i += numThreads) {
    column = i / height;
    row = i % height;
    source_pos = height * (int)indices1[column] + row;
    target_pos = height * (int)indices2[column] + row;
    temp = source[source_pos];
    source[source_pos] = target[target_pos];
    target[target_pos] = temp;
  }
}

__global__ void kShuffleColumns(float* source, float* target, float* indices, int width, int height){
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  float temp1, temp2;
  unsigned int column, row, pos1, pos2;

  for (unsigned int i = idx; i < height * ((width+1) / 2); i += numThreads) {
    column = 2 * (i / height);
    row = i % height;
    if (column + 1 >= width) {
      pos1 = height * (int)indices[column] + row;
      target[pos1] = source[pos1];
    } else {
      pos1 = height * (int)indices[column] + row;
      pos2 = height * (int)indices[column + 1] + row;
      temp1 = source[pos1];
      temp2 = source[pos2];
      target[pos2] = temp1;
      target[pos1] = temp2;
    }
  }
}


__global__ void kSetSelectedRows(float* target, float* source, float* indices, int nRowIs, int nCols, int nTargetRows){
  __shared__ int targetRowIndices[32];
  const int startSourceRowI = blockIdx.x * 32;
  const int tid = threadIdx.x;
  const int localNRowIs = min(32, nRowIs-startSourceRowI);

  // cooperatively load 32 row indices
  if (tid < localNRowIs){
    targetRowIndices[tid] = int(indices[startSourceRowI + tid]);
    if (targetRowIndices[tid]<0)
      targetRowIndices[tid] += nTargetRows;
    if (targetRowIndices[tid]<0 || targetRowIndices[tid]>=nTargetRows)
      targetRowIndices[tid] = -1;
  }
  __syncthreads();

  // copy 32 rows
  for (int i=0; i<localNRowIs; i++){
    const int sourceRowI = startSourceRowI + i, targetRowI = targetRowIndices[i];
    for (int colI=tid; colI<nCols; colI+=32)
      target[targetRowI * nCols + colI] = targetRowI==-1 ? (1.0/0.0 -1.0/0.0) : source[sourceRowI * nCols + colI];
  }
}

__global__ void kBlockify(float* source, float* target, int numdims, int blocksize) {
  const unsigned int idx = threadIdx.x;
  const unsigned int numThreads = blockDim.x;
  const int off = blockIdx.x * numdims;

  for (unsigned int target_ind = idx; target_ind < numdims; target_ind += numThreads) {
    const int block = target_ind / blocksize;
    target[off + target_ind] = source[off + block * blocksize];
  }
}

__global__ void kGenerateTranslationsBigVarOff(float* source, float* target, float* off_x_arr, float* off_y_arr, int source_w, int target_w, int num_channels) {
  const unsigned int idx = threadIdx.x;
  const unsigned int numThreads = blockDim.x;

  int target_x, target_y;
  int pad = (source_w - target_w)/2;
  int target_tile_size = target_w * target_w;
  int source_tile_size = source_w * source_w;

  int off_x = off_x_arr[blockIdx.x];
  int off_y = off_y_arr[blockIdx.x];
  int target_off = blockIdx.x * target_tile_size;
  int source_off = blockIdx.x * source_tile_size + (pad + off_x) * source_w + (pad + off_y);

  for (unsigned int target_ind = idx; target_ind < target_tile_size; target_ind += numThreads) {
    target_x = target_ind / target_w;
    target_y = target_ind - target_x * target_w;

    for (unsigned int ch = 0; ch < num_channels; ch += 1) {
      target[num_channels*(target_off + target_x * target_w + target_y) + ch] = source[num_channels*(source_off + target_x * source_w + target_y) + ch];
    }
  }
}

__global__ void kSoftMaxGrad(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width * height; i += numThreads) {
    target[i] = mat[i] - ((int)labels[i / height] == i % height ? 1 : 0);
  }
}

__global__ void kSoftMaxGradRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width*height; i += numThreads) {
    target[i] = mat[i] - ((int)labels[i % height] == i / height ? 1 : 0);
  }
}

__global__ void kHingeQuadraticRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float margin) {
  int image_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (image_id < height) {
    mat += image_id;
    target += image_id;
    const int correct_label = (int)labels[image_id];
    const float correct_label_score = mat[correct_label * height];
    float sum = 0;
    for (unsigned int i = 0; i < width; i++) {
      float diff = margin + mat[i*height] - correct_label_score;
      float grad = (diff > 0) ? diff : 0;
      target[i*height] = (i == correct_label) ? 0 : grad;
      sum             += (i == correct_label) ? 0 : grad;
    }
    target[correct_label * height] = -sum;
  }
}

__global__ void kHingeLinearRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float margin) {
  int image_id = blockIdx.x * blockDim.x + threadIdx.x;
  if (image_id < height) {
    mat += image_id;
    target += image_id;
    const int correct_label = (int)labels[image_id];
    const float correct_label_score = mat[correct_label * height];
    float sum = 0;
    for (unsigned int i = 0; i < width; i++) {
      float diff = margin + mat[i*height] - correct_label_score;
      float grad = (diff > 0) ? 1 : 0;
      target[i*height] = (i == correct_label) ? 0 : grad;
      sum             += (i == correct_label) ? 0 : grad;
    }
    target[correct_label * height] = -sum;
  }
}

__global__ void kSoftMaxGradCLS(float* mat, int* labels, float* indices, float* target, unsigned int width, unsigned int height) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width*height; i += numThreads) {
    target[i] = mat[i] - (labels[(int)indices[i % height]] == i / height ? 1 : 0);
  }
}

__global__ void kSoftMaxCrossEntropy(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float tiny) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width; i += numThreads) {
    target[i] = -__logf(mat[height * i + (int)labels[i]] + tiny);
  }
}

__global__ void kSoftMaxCrossEntropyRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height, float tiny) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < height; i += numThreads) {
    target[i] = -__logf(mat[height * (int)labels[i] + i] + tiny);
  }
}


__global__ void kSoftMaxCorrect(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
  __shared__ float max_vals[32];
  __shared__ unsigned int max_val_args[32];
  float cur_max = -FLT_MAX;
  unsigned int cur_argmax = 0;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) {
        cur_max = val;
        cur_argmax = i;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;
    __syncthreads();
    if (threadIdx.x == 0) {
      cur_max = -FLT_MAX;
      cur_argmax = 0;
      for (unsigned int i = 0; i < blockDim.x; i++)
        if (max_vals[i] > cur_max) {
          cur_max = max_vals[i];
          cur_argmax = max_val_args[i];
        }   
      target[column] = (cur_argmax == (int)labels[column]) ? 1 : 0;
    }
  }
}

__global__ void kSoftMaxCorrectRowMajor(float* mat, float* labels, float* target, unsigned int width, unsigned int height) {
  __shared__ float max_vals[32];
  __shared__ unsigned int max_val_args[32];
  float cur_max = -FLT_MAX;
  unsigned int cur_argmax = 0;
  float val = 0;
  const int row = gridDim.x * blockIdx.y + blockIdx.x;
  if (row < height) {
    float *cur_data = &mat[row] ; 
    for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) {
      val = cur_data[i * height];
      if (val > cur_max) {
        cur_max = val;
        cur_argmax = i;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;
    __syncthreads();
    if (threadIdx.x == 0) {
      cur_max = -FLT_MAX;
      cur_argmax = 0;
      for (unsigned int i = 0; i < blockDim.x; i++)
        if (max_vals[i] > cur_max) {
          cur_max = max_vals[i];
          cur_argmax = max_val_args[i];
        }   
      target[row] = (cur_argmax == (int)labels[row]) ? 1 : 0;
    }
  }
}

__global__ void kSoftMaxCorrectCLS(float* mat, int* labels, float* indices, float* target, unsigned int width, unsigned int height) {
  __shared__ float max_vals[32];
  __shared__ unsigned int max_val_args[32];
  float cur_max = -FLT_MAX;
  unsigned int cur_argmax = 0;
  float val = 0;
  const int row = gridDim.x * blockIdx.y + blockIdx.x;
  if (row < height) {
    float *cur_data = &mat[row] ; 
    for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) {
      val = cur_data[i * height];
      if (val > cur_max) {
        cur_max = val;
        cur_argmax = i;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;
    __syncthreads();
    if (threadIdx.x == 0) {
      cur_max = -FLT_MAX;
      cur_argmax = 0;
      for (unsigned int i = 0; i < blockDim.x; i++)
        if (max_vals[i] > cur_max) {
          cur_max = max_vals[i];
          cur_argmax = max_val_args[i];
        }   
      target[row] = (cur_argmax == labels[(int)indices[row]]) ? 1 : 0;
    }
  }
}





__global__ void kSoftMax(float* mat, float* target, unsigned int width, unsigned int height) {
  extern __shared__ float max_vals[] ;
  float cur_max = -FLT_MAX;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    max_vals[threadIdx.x]=-FLT_MAX;
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) {
        cur_max = val;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    reduceToMax(max_vals, threadIdx.x);
    __syncthreads();
    cur_max = max_vals[0] ; 
    __syncthreads();
    val = 0;
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val += __expf(cur_data[i]-cur_max);
    }
    max_vals[threadIdx.x] = val;
    reduceToSumLocal(max_vals, threadIdx.x);
    __syncthreads();
    float norm = max_vals[0] ; 
    float *cur_target = &target[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      cur_target[i] = __expf(cur_data[i]-cur_max) / norm ;
    }
  }
}

__global__ void kSoftMaxOverwrite(float* mat, unsigned int width, unsigned int height) {
  extern __shared__ float max_vals[] ;
  float cur_max = -FLT_MAX;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    max_vals[threadIdx.x]=-FLT_MAX;
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) {
        cur_max = val;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    reduceToMax(max_vals, threadIdx.x);
    __syncthreads();
    cur_max = max_vals[0] ;
    __syncthreads();
    val = 0;
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      cur_data[i] = __expf(cur_data[i]-cur_max);
      val += cur_data[i];
    }
    max_vals[threadIdx.x] = val;
    reduceToSumLocal(max_vals, threadIdx.x);
    __syncthreads();
    float norm = max_vals[0] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      cur_data[i] /= norm;
    }
  }
}

__global__ void kSoftMaxRowMajor(float* mat, unsigned int width, unsigned int height) {
  extern __shared__ float max_vals[] ;
  float cur_max = -FLT_MAX;
  float val = 0;
  const int row = gridDim.x * blockIdx.y + blockIdx.x;
  if (row < height) {
    float *cur_data = &mat[row] ; 
    max_vals[threadIdx.x]=-FLT_MAX;
    for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) {
      val = cur_data[i * height];
      if (val > cur_max) {
        cur_max = val;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    reduceToMax(max_vals, threadIdx.x);
    __syncthreads();
    cur_max = max_vals[0] ;
    __syncthreads();
    val = 0;
    for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) {
      cur_data[i * height] = __expf(cur_data[i * height]-cur_max);
      val += cur_data[i * height];
    }
    max_vals[threadIdx.x] = val;
    reduceToSumLocal(max_vals, threadIdx.x);
    __syncthreads();
    float norm = max_vals[0] ; 
    for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) {
      cur_data[i * height] /= norm;
    }
  }
}

__global__ void kChooseMaxAndAccumulate(float* mat, float* target, unsigned int width, unsigned int height) {
  __shared__ float max_vals[32];
  __shared__ unsigned int max_val_args[32];
  float cur_max = -FLT_MAX;
  unsigned int cur_argmax = 0;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    float *target_data = &target[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) {
        cur_max = val;
        cur_argmax = i;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;
    __syncthreads();
    if (threadIdx.x == 0) {
      cur_max = -FLT_MAX;
      cur_argmax = 0;
      for (unsigned int i = 0; i < blockDim.x; i++)
        if (max_vals[i] > cur_max) {
          cur_max = max_vals[i];
          cur_argmax = max_val_args[i];
        }   
      target_data[cur_argmax] += 1;
    }
  }
}
__global__ void kChooseMaxColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
  __shared__ float max_vals[32];
  __shared__ unsigned int max_val_args[32];
  float cur_max = -FLT_MAX;
  unsigned int cur_argmax = 0;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    float *target_data = &target[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      target[i] = 0;
      if (val > cur_max) {
        cur_max = val;
        cur_argmax = i;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;
    __syncthreads();
    if (threadIdx.x == 0) {
      cur_max = -FLT_MAX;
      cur_argmax = 0;
      for (unsigned int i = 0; i < blockDim.x; i++)
        if (max_vals[i] > cur_max) {
          cur_max = max_vals[i];
          cur_argmax = max_val_args[i];
        }   
      target_data[cur_argmax] = 1;
    }
  }
}

__global__ void kMaxColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
  extern __shared__ float max_vals[] ;
  float cur_max = -FLT_MAX;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) cur_max = val;
    }
    max_vals[threadIdx.x] = cur_max;
    reduceToMax(max_vals, threadIdx.x);
    __syncthreads();
    if (threadIdx.x == 0) target[column] = max_vals[0];
  }
}

__global__ void kArgMaxColumnwise(float* mat, float* target, unsigned int width, unsigned int height) {
  __shared__ float max_vals[32];
  __shared__ unsigned int max_val_args[32];
  float cur_max = -FLT_MAX;
  unsigned int cur_argmax = 0;
  float val = 0;
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float *cur_data = &mat[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      val = cur_data[i];
      if (val > cur_max) {
        cur_max = val;
        cur_argmax = i;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;
    __syncthreads();
    if (threadIdx.x == 0) {
      cur_max = -FLT_MAX;
      cur_argmax = 0;
      for (unsigned int i = 0; i < blockDim.x; i++)
        if (max_vals[i] > cur_max) {
          cur_max = max_vals[i];
          cur_argmax = max_val_args[i];
        }   
      target[column] = cur_argmax;
    }
  }
}

__global__ void kSqSumColumnwise(float* mat, float* target, unsigned int width, unsigned int height, float mult, float p) {
  extern __shared__ float sum_vals[];
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float cur_sum = 0;
    float *cur_data = &mat[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      cur_sum += cur_data[i] * cur_data[i];
    }
    sum_vals[threadIdx.x] = cur_sum;
    reduceToSumLocal(sum_vals, threadIdx.x);
    __syncthreads();
    if (threadIdx.x == 0) target[column] = p * target[column] + mult * sum_vals[0];
  }
}

__global__ void kSumColumnwise(float* mat, float* target, unsigned int width, unsigned int height, float mult, float p) {
  extern __shared__ float sum_vals[];
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float cur_sum = 0;
    float *cur_data = &mat[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      cur_sum += cur_data[i];
    }
    sum_vals[threadIdx.x] = cur_sum;
    reduceToSumLocal(sum_vals, threadIdx.x);
    __syncthreads();
    if (threadIdx.x == 0) target[column] = p * target[column] + mult * sum_vals[0];
  }
}

__global__ void kSqSumRowwise(float* mat, float* target, unsigned int width, unsigned int height, float mult, float p) {
  extern __shared__ float sum_vals[];
  const int row = gridDim.x * blockIdx.y + blockIdx.x;
  if (row < height) {
    float cur_sum = 0;
    float *cur_data = &mat[row] ; 
    for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) {
      cur_sum += cur_data[i * height] * cur_data[i * height];
    }
    sum_vals[threadIdx.x] = cur_sum;
    reduceToSumLocal(sum_vals, threadIdx.x);
    __syncthreads();
    if (threadIdx.x == 0) target[row] = p * target[row] + mult * sum_vals[0];
  }
}
__global__ void kSumRowwise(float* mat, float* target, unsigned int width, unsigned int height, float mult, float p) {
  extern __shared__ float sum_vals[];
  const int row = gridDim.x * blockIdx.y + blockIdx.x;
  if (row < height) {
    float cur_sum = 0;
    float *cur_data = &mat[row] ; 
    for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) {
      cur_sum += cur_data[i * height];
    }
    sum_vals[threadIdx.x] = cur_sum;
    reduceToSumLocal(sum_vals, threadIdx.x);
    __syncthreads();
    if (threadIdx.x == 0) target[row] = p * target[row] + mult * sum_vals[0];
  }
}


__global__ void kNormLimitColumnwise(float* mat, float* target, float norm, unsigned int width, unsigned int height, int constraint) {
  extern __shared__ float sum_vals[];
  const int column = gridDim.x * blockIdx.y + blockIdx.x;
  if (column < width) {
    float cur_sum = 0;
    float *cur_data = &mat[column * height] ; 
    float *target_data = &target[column * height] ; 
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      cur_sum += cur_data[i] * cur_data[i];
    }
    sum_vals[threadIdx.x] = cur_sum;
    reduceToSumLocal(sum_vals, threadIdx.x);
    __syncthreads();
    cur_sum = sqrt(sum_vals[0]);
    cur_sum = (constraint == 1 || cur_sum > norm) ? (norm / cur_sum) : 1;
    for (unsigned int i = threadIdx.x; i < height; i += blockDim.x) {
      target_data[i] = cur_data[i] * cur_sum;
    }
    __syncthreads();
  }
}

__global__ void kNormLimitRowwise(float* mat, float* target, float norm, unsigned int width, unsigned int height, int constraint) {
  extern __shared__ float sum_vals[];
  const int row = gridDim.x * blockIdx.y + blockIdx.x;
  if (row < height) {
    float cur_sum = 0;
    float *cur_data = &mat[row] ; 
    float *target_data = &target[row] ; 
    for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) {
      cur_sum += cur_data[i * height] * cur_data[i * height];
    }
    sum_vals[threadIdx.x] = cur_sum;
    reduceToSumLocal(sum_vals, threadIdx.x);
    __syncthreads();
    cur_sum = sqrt(sum_vals[0]);
    cur_sum = (constraint == 1 || cur_sum > norm) ? (norm / cur_sum) : 1;
    for (unsigned int i = threadIdx.x; i < width; i += blockDim.x) {
      target_data[i * height] = cur_data[i * height] * cur_sum;
    }
    __syncthreads();
  }
}

__global__ void kExpand(float* source, float* indices, float* target, int height, int width, int target_width){
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < target_width*height; i += numThreads) {
    const int pos = height * (int)indices[i / height] + i % height;
    target[i] = (pos < height * width)? source[pos] : 1.0/0.0 - 1.0/0.0;
  }
}


__global__ void kExpandAndAdd(float* source, float* mat, float* indices, float* target, int width, int height, float mult, int width2){
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < width*height; i += numThreads) {
    const int pos = height * (int)indices[i / height] + i % height;
    target[i] = (pos < height * width2)? source[i] + mult * mat[pos] : 1.0/0.0 - 1.0/0.0;
  }
}

__global__ void kAccumulateColumns(float* mat, float* indices, float* target, int mat_width, int target_width, int height, float mult, int avg){
  const int row = gridDim.x * blockIdx.y + blockIdx.x;
  const int column = threadIdx.x;
  if (row < height && column < target_width) {
    float cur_sum = 0.0;
    unsigned int count = 0;
    for (unsigned int i = 0; i < mat_width; i ++) {
      count += ((int)indices[i] == column) ? 1 : 0 ;
      cur_sum += ((int)indices[i] == column) ? mat[row + i * height] : 0 ;
    }
    target[row + height * column] = mult * cur_sum / ((avg == 1 && count > 0) ? count : 1);
  }
}

__global__ void kExtractPatches(float* images, float* patches, float* indices, float* width_offset, float* height_offset, int num_images, int img_width, int img_height, int patch_width, int patch_height, int num_colors) {
  const unsigned long idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned long numThreads = blockDim.x * gridDim.x;
  const unsigned long total_pixels = patch_width * patch_height * num_colors * num_images;
  unsigned long ind, pos;
  unsigned long image_id, dest_row, dest_col, color, source_row, source_col;
  for (unsigned long i = idx; i < total_pixels; i += numThreads) {
    ind = i;
    image_id = ind % num_images; ind /= num_images;
    dest_col = ind % patch_width; ind /= patch_width;
    dest_row = ind % patch_height; ind /= patch_height;
    color = ind % num_colors;

    source_row = int(height_offset[image_id]) + dest_row;
    source_col = int(width_offset[image_id]) + dest_col;
    pos = img_width * img_height * num_colors * (int)indices[image_id] + img_width * img_height * color + img_width * source_row + source_col;
    patches[i] = images[pos];
  }
}

__global__ void kExtractPatches2(float* images, float* patches, float* width_offset, float* height_offset, float* flip, int num_images, int img_width, int img_height, int patch_width, int patch_height, int num_colors) {
  int image_id = blockIdx.z;
  int dest_col = blockIdx.x * blockDim.x + threadIdx.x;
  int dest_row = blockIdx.y * blockDim.y + threadIdx.y;
  int color = threadIdx.z;

  if (dest_col < patch_width && dest_row < patch_height) {
    int source_row = int(height_offset[image_id]) + dest_row;
    int source_col = int(width_offset[image_id]) + dest_col;
    source_col = (flip[image_id] > 0.5) ? (img_width - source_col - 1) : source_col;
    unsigned long dest_index = image_id + num_images * (dest_col  + patch_width * (dest_row + patch_height * color));
    unsigned long source_index = source_col + img_width * (source_row + img_height * (color + num_colors * image_id));
    patches[dest_index] = images[source_index];
  }
}

__global__ void kRectifyBoundingBox(
    float* boxes, float* width_offset, float* height_offset, float* flip,
    int num_images, int patch_width, int patch_height, int num_locs) {

  for (int loc_id = blockIdx.x; loc_id < num_locs; loc_id += gridDim.x) {
    float *xmin_block = boxes + num_images * loc_id,
          *ymin_block = boxes + num_images * (loc_id + num_locs),
          *xmax_block = boxes + num_images * (loc_id + num_locs * 2),
          *ymax_block = boxes + num_images * (loc_id + num_locs * 3);

    for (int image_id = threadIdx.x; image_id < num_images; image_id += blockDim.x) {
      float xmin = (flip[image_id] > 0.5) ? (256.0/patch_width - xmax_block[image_id]) : xmin_block[image_id],
            xmax = (flip[image_id] > 0.5) ? (256.0/patch_width - xmin_block[image_id]) : xmax_block[image_id],
            ymin = ymin_block[image_id],
            ymax = ymax_block[image_id],
            wo = width_offset[image_id],
            ho = height_offset[image_id];

      xmin_block[image_id] = xmin - wo / patch_width;
      xmax_block[image_id] = xmax - wo / patch_width;

      ymin_block[image_id] = ymin - ho / patch_height;
      ymax_block[image_id] = ymax - ho / patch_height;
    }
  }
}

__global__ void kAdagrad(float *w, float *grad, float *sum_grad_sq, int len, float decay, float epsilon) {
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  for (unsigned int i = idx; i < len; i += numThreads) {
    sum_grad_sq[i] = decay * sum_grad_sq[i] + grad[i] * grad[i];
    w[i] += epsilon * grad[i] / (0.00001 + sqrtf(sum_grad_sq[i]));
  }
}

__global__ void kBoundingBoxLogisticGrad(
    float* mat, int* bbox, int* label, int* seg, float* indices, float* width_offset, float* height_offset,
    int size, int width, int height, int depth, float scale_width, float scale_height, float* grad) {
  const int color = blockIdx.z;
  /*
  const int numXBlocksPerImage = DIVUP(width, blockDim.x);
  const int image_id = blockIdx.x / numXBlocksPerImage;
  const int col = (blockIdx.x % numXBlocksPerImage) * blockDim.x + threadIdx.x;
  const int row = blockIdx.y * blockDim.y + threadIdx.y;
  */
  const int image_id = threadIdx.x;
  const int col = blockIdx.x;
  const int row = blockIdx.y;

  int num_bboxes = 0, num_bboxes_of_this_depth = 0, num_bboxes_of_this_depth_inside = 0;
  if (col < width && row < height && image_id < size && color < depth) {
    int src_image_id = (int)indices[image_id];
    int src_col = (int)(scale_width * col);
    int src_row = (int)(scale_height * row);

    int start = seg[src_image_id];
    int end = seg[src_image_id + 1];
    int x1, y1, x2, y2, l, inside;
    for (int box_id = start; box_id < end; box_id++) {
      l = label[box_id];
      x1 = bbox[box_id << 2] - width_offset[image_id];
      y1 = bbox[(box_id << 2) + 1] - height_offset[image_id];
      x2 = bbox[(box_id << 2) + 2] - width_offset[image_id];
      y2 = bbox[(box_id << 2) + 3] - height_offset[image_id];
      inside = (src_col >= x1 && src_col <= x2 && src_row >= y1 && src_row <= y2) ? 1:0;
      num_bboxes += inside;
      num_bboxes_of_this_depth += (l == color) ? 1: 0;
      num_bboxes_of_this_depth_inside += (inside == 1 && l == color) ? 1: 0;
    }
  }
  unsigned long i = image_id + size * (col + width * (row + height * color));
  __syncthreads();
  if (col < width && row < height && image_id < size && color < depth) {
    if (num_bboxes > 0) {
      grad[i] = (num_bboxes_of_this_depth_inside > 0) ? (mat[i] - 1) : 0;
    } else {
      grad[i] = (num_bboxes_of_this_depth > 0) ? mat[i] : 0;
    }
  }
}

__global__ void kLogisticCorrectBoundingBox(
    float* mat, int* bbox, int* label, int* seg, float* indices,
    float* width_offset, float* height_offset, int size, int width, int height,
    int depth, float scale_width, float scale_height, float* target, float cutoff) {
  const int color = blockIdx.z;
  const int numXBlocksPerImage = DIVUP(width, blockDim.x);
  const int image_id = blockIdx.x / numXBlocksPerImage;
  const int col = (blockIdx.x % numXBlocksPerImage) * blockDim.x + threadIdx.x;
  const int row = blockIdx.y * blockDim.y + threadIdx.y;

  if (col < width && row < height && image_id < size && color < depth) {
    int src_image_id = (int)indices[image_id];
    int src_col = (int)(scale_width * col);
    int src_row = (int)(scale_height * row);

    int start = seg[src_image_id];
    int end = seg[src_image_id + 1];
    int x1, y1, x2, y2, l, inside;
    int num_bboxes = 0, num_bboxes_of_this_depth = 0, num_bboxes_of_this_depth_inside = 0;
    for (int box_id = start; box_id < end; box_id++) {
      l = label[box_id];
      x1 = bbox[box_id << 2] - width_offset[image_id];
      y1 = bbox[(box_id << 2) + 1] - height_offset[image_id];
      x2 = bbox[(box_id << 2) + 2] - width_offset[image_id];
      y2 = bbox[(box_id << 2) + 3] - height_offset[image_id];
      inside = (src_col >= x1 && src_col <= x2 && src_row >= y1 && src_row <= y2) ? 1:0;
      num_bboxes += inside;
      num_bboxes_of_this_depth += (l == color) ? 1: 0;
      num_bboxes_of_this_depth_inside += (inside == 1 && l == color) ? 1: 0;
    }
    unsigned long i = image_id + size * (col + width * (row + height * color));
    if (num_bboxes > 0) {
      target[i] = (num_bboxes_of_this_depth_inside > 0 && mat[i] >= cutoff) ? 1 : 0;
    } else {
      target[i] = (num_bboxes_of_this_depth > 0 && mat[i] < cutoff) ? 1 : 0;
    }
  }
}

__global__ void kBoundingBoxSoftMaxGrad(
    float* mat, int* bbox, int* label, int* seg, float* indices, float* width_offset, float* height_offset,
    int size, int width, int height, int depth, float scale_width, float scale_height, float* grad) {
  const unsigned int len = width * height * depth * size;
  const unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int numThreads = blockDim.x * gridDim.x;
  int ind, image_id, source_depth, x1, y1, x2, y2, start,
      end, src_image_id, num_bboxes, num_bboxes_of_this_depth, box_id, inside;
  float source_x, source_y;
  for (unsigned int i = idx; i < len; i += numThreads) {
    ind = i;
    image_id = ind % size; ind /= size;
    source_x = scale_width * (ind % width); ind /= width;
    source_y = scale_height * (ind % height); ind /= height;
    source_depth = ind % depth;
    src_image_id = (int)indices[image_id];

    start = seg[src_image_id];
    end = seg[src_image_id + 1];
    num_bboxes = 0;
    num_bboxes_of_this_depth = 0;
    for (box_id = start; box_id < end; box_id++) {
      x1 = bbox[box_id << 2] - width_offset[image_id];
      y1 = bbox[(box_id << 2) + 1] - height_offset[image_id];
      x2 = bbox[(box_id << 2) + 2] - width_offset[image_id];
      y2 = bbox[(box_id << 2) + 3] - height_offset[image_id];
      inside = (source_x >= x1 && source_x <= x2 && source_y >= y1 && source_y <= y2) ? 1:0;
      num_bboxes += inside;
      num_bboxes_of_this_depth += (inside == 1 && label[box_id] == source_depth) ? 1: 0;
    }
    grad[i] = mat[i] - ((num_bboxes > 0) ? ((float)num_bboxes_of_this_depth / num_bboxes) : (source_depth == 0 ? 1:0));
  }
}

__global__ void kSoftMaxCorrectBoundingBox(
    float* mat, int* bbox, int* label, int* seg, float* indices,
    float* width_offset, float* height_offset, int size, int width, int height,
    int depth, float scale_width, float scale_height, float* target) {
  const int row = gridDim.x * blockIdx.y + blockIdx.x;
  const int num_pixels = size * width * height;
  if (row < num_pixels) {
    __shared__ float max_vals[32];
    __shared__ unsigned int max_val_args[32];
    float cur_max = -FLT_MAX;
    unsigned int cur_argmax = 0;
    float val = 0;

    float *cur_data = &mat[row] ; 
    for (unsigned int i = threadIdx.x; i < depth; i += blockDim.x) {
      val = cur_data[i * num_pixels];
      if (val > cur_max) {
        cur_max = val;
        cur_argmax = i;
      }
    }
    max_vals[threadIdx.x] = cur_max;
    max_val_args[threadIdx.x] = cur_argmax;
    __syncthreads();
    if (threadIdx.x == 0) {
      cur_max = -FLT_MAX;
      cur_argmax = 0;
      for (unsigned int i = 0; i < blockDim.x; i++)
        if (max_vals[i] > cur_max) {
          cur_max = max_vals[i];
          cur_argmax = max_val_args[i];
        }
      int ind, image_id, src_image_id, x1, y1, x2, y2, start,
        end, num_bboxes, correct, box_id, inside;
      float source_x, source_y;
      ind = row;
      image_id = ind % size; ind /= size;
      source_x = scale_width * (ind % width); ind /= width;
      source_y = scale_height * (ind % height); ind /= height;
      src_image_id = (int)indices[image_id];
      start = seg[src_image_id];
      end = seg[src_image_id + 1];
      num_bboxes = 0;
      correct = 0;
      for (box_id = start; box_id < end; box_id++) {
        x1 = bbox[box_id << 2] - width_offset[image_id];
        y1 = bbox[(box_id << 2) + 1] - height_offset[image_id];
        x2 = bbox[(box_id << 2) + 2] - width_offset[image_id];
        y2 = bbox[(box_id << 2) + 3] - height_offset[image_id];
        inside = (source_x >= x1 && source_x <= x2 && source_y >= y1 && source_y <= y2) ? 1:0;
        num_bboxes += inside;
        correct += (inside == 1 && cur_argmax == label[box_id]) ? 1 : 0;
      }
      target[row] = (num_bboxes > 0) ? ((correct > 0) ? 1 : 0) : ((cur_argmax == 0) ? 1: 0);
    }
  }
}
